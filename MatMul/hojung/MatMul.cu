#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "DS_timer.cpp"
#define col_size 1024
#define k_size 512
#define row_size 1024
#define divide 32


__global__ void MatMul(float* _a, float* _b, float* _c){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int index = row * blockDim.x* gridDim.x + col;
    _c[index] = 0;
    for(int k = 0; k < k_size; k++){
        _c[index] += _a[row*k_size+k]*_b[col+k*col_size];  
      /*  if(index == 32)
          printf("GPU [%d] is a[%d] %f X b[%d] %f\n",k,row*blockDim.x+k,_a[row*blockDim.x+k],col+k*blockDim.x,_b[col+k*blockDim.x]); */
    }
   // printf("%d is %d\n",index, _c[index]);
}

int main(void){
    float *a, *b, *c, *d;
    float *d_a, *d_b, *d_c;
    int a_size = col_size*k_size;
    int b_size = row_size*k_size;
    dim3 dimBlock(col_size/divide, row_size/divide);
    dim3 dimGrid(divide,divide);

    DS_timer timer(5);
    timer.setTimerName(0, "CUDA Total");
    timer.setTimerName(1, "Computation(Kernel)");
    timer.setTimerName(2, "Data Trans. : Host -> Device");
    timer.setTimerName(3, "Data Trans. : Device -> Host");
    timer.setTimerName(4, "VectorSum on Host");
    timer.initTimers();

    a = new float[a_size];
    memset(a, 0, sizeof(float)*a_size);
    b = new float[b_size];
    memset(b, 0, sizeof(float)*b_size);
    c = new float[row_size*col_size];
    memset(b, 0, sizeof(float)*row_size*col_size);
    d = new float[row_size*col_size];
    memset(b, 0, sizeof(float)*row_size*col_size);

    for(int i = 0; i<a_size;i++){
        a[i] = rand() % 10;
    }
    for(int i = 0; i<b_size;i++){
        b[i] = rand() % 10;
    }

    hipMalloc(&d_a, sizeof(float)*a_size);
    hipMalloc(&d_b, sizeof(float)*b_size);
    hipMalloc(&d_c, sizeof(float)*row_size*col_size);

    timer.onTimer(0);
    timer.onTimer(2);
    hipMemcpy(d_a, a, sizeof(float)*a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*b_size, hipMemcpyHostToDevice);
    timer.offTimer(2);
    timer.onTimer(1);
    MatMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    hipDeviceSynchronize(); // synchronization function
	  timer.offTimer(1);

    timer.onTimer(3);
    hipMemcpy(c, d_c, sizeof(float)*col_size*row_size, hipMemcpyDeviceToHost);
	  timer.offTimer(3);
    timer.offTimer(0);
    bool isCorrect = true;

    timer.onTimer(4);
    for(int i = 0; i < row_size; i++){
        for(int j = 0; j < col_size; j++){
            for(int k = 0; k < k_size; k++){
                d[i*col_size+j] += a[i*k_size+k]*b[j+k*col_size];
            /*    if((i*col_size+j)== 32)
                    printf("CPU [%d] is a[%d] %f X b[%d] %f\n",k,i*k_size+k,a[i*k_size+k],j+k*col_size,b[j+k*col_size]); */
            }
        }
    }
    timer.offTimer(4); timer.printTimer();
    for(int i = 0; i < col_size*row_size; i++){
        if(d[i] != c[i]){
            printf("[%d] result is not matched, (%f, %f)\n",i,d[i],c[i]);
            isCorrect = false;
        }
    }

    if(isCorrect){
        printf("Result is same");
    }

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    delete [] a; delete [] b; delete [] c; delete [] d;
    return 0;
}
